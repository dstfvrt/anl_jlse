
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>


#define N 1000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation. Checksum: 499495.
 * Runtime values from gpu_a100 JLSE machine. 
 * Approx. runtimes: Total: 76-82ms. Mem alloc: 76-82ms.
 * Reduction: 0.02ms. 
 */

__global__ void reduce_GPU(unsigned* d) {
    /* Shared memory */
    extern __shared__ unsigned sdata[];

    /* load into shared memory */
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d[i];
    __syncthreads();

    /* reduction */
    for (unsigned s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    if (tid==0) d[0] = sdata[0];
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

/* CPU timing functions */
int main(int argc, char **argv) {

    unsigned* h;
    unsigned* d;
    unsigned* result;
    unsigned nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    result = (unsigned *)malloc(nBytes);
    dense(h);

    /* Timing variables */
    struct timeval etstart, etstop;
    struct timezone tzdummy;
    clock_t etstart2, etstop2;
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;

    hipMalloc(&d, nBytes);
    hipMemcpy(d, h, nBytes, hipMemcpyHostToDevice);

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

    reduce_GPU<<<(N+1023) / 1024, 1024, nBytes>>>(d);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;
    printf("\nElapsed time = %g ms.\n",
           (float)(usecstop - usecstart)/(float)1000);

    hipMemcpy(result, d, nBytes, hipMemcpyDeviceToHost);
    printf("Checksum: %u\n", result[0]);

    exit(0);
}
