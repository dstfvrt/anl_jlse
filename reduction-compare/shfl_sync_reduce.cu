#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__inline__ __device__
float warpReduceRedux(float val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down(val, offset);
    return val;
}

__inline__ __device__
float blockReduceRedux(float val) {
    static __shared__ int shared[32];
    int lane = threadIdx.x%32;
    int wid = threadIdx.x / 32;
    val = warpReduceRedux(val);

    if (lane == 0) shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x<blockDim.x / 32) ? shared[lane] : int(0);
    if (wid == 0) val = warpReduceRedux(val);

    return val;
}

__global__ void device_redux_sync_kernel(float *in, float* out, int N) {
    float sum = int(0);
    for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<N; i += blockDim.x*gridDim.x) {
        sum += in[i];
    }
    sum = blockReduceRedux(sum);
    if (threadIdx.x == 0)
        out[blockIdx.x] = sum;
}

void device_reduce_redux_sync(float *in, float* out, int N) {
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    int blocks = N / maxThreadsPerBlock;
    // Begin device execution -- reduce to 1 value per block
    device_redux_sync_kernel<<<blocks, threads >>>(in, out, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // Begin device execution -- reduce across blocks to scalar
    device_redux_sync_kernel<<<1, 1024 >>>(out, out, blocks);
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
}

__inline__ __device__
float warpReduceShfl(float val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down(val, offset);
    return val;
}

__inline__ __device__
float blockReduceShfl(float val) {
    static __shared__ int shared[32];
    int lane = threadIdx.x%32;
    int wid = threadIdx.x / 32;
    val = warpReduceShfl(val);

    if (lane == 0) shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x<blockDim.x / 32) ? shared[lane] : int(0);
    if (wid == 0) val = warpReduceShfl(val);

    return val;
}

__global__ void device_shfl_down_kernel(float *in, float* out, int N) {
    float sum = int(0);
    for (int i = blockIdx.x*blockDim.x + threadIdx.x; i<N; i += blockDim.x*gridDim.x) {
        sum += in[i];
    }
    sum = blockReduceShfl(sum);
    if (threadIdx.x == 0)
        out[blockIdx.x] = sum;
}

void device_reduce_shfl_down(float *in, float* out, int N) {
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    int blocks = N / maxThreadsPerBlock;
    // Begin device execution -- reduce to 1 value per block
    device_shfl_down_kernel<<<blocks, threads >>>(in, out, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    // Begin device execution -- reduce across blocks to scalar
    device_shfl_down_kernel<<<1, 1024 >>>(out, out, blocks);
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
}


int main()
{
    const int ARRAY_SIZE = 2048;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float sum = 0.0f;
    for (int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [-1.0f, 1.0f]
        h_in[i] = i;
        sum += h_in[i];
    }

    // declare GPU memory pointers
    float * d_in, *d_intermediate, *d_out;

    // allocate GPU memory
    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_intermediate, ARRAY_BYTES); // overallocated
    hipMalloc((void **)&d_out, sizeof(float));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    int whichKernel = 2;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // launch the kernel
    hipProfilerStart();
    switch (whichKernel) {
    case 1:
        printf("Running reduce with shuffle instruction\n");
        hipEventRecord(start, 0);
            device_reduce_stable(d_in, d_out, ARRAY_SIZE);
        hipEventRecord(stop, 0);
        break;
    case 2:
        printf("Running reduce with redux instruction\n");
        hipEventRecord(start, 0);
            device_reduce_stable(d_in, d_out, ARRAY_SIZE);
        hipEventRecord(stop, 0);
        break;
    default:
        fprintf(stderr, "error: ran no kernel\n");
        exit(EXIT_FAILURE);
    }
    hipProfilerStop();
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    elapsedTime /= 100.0f;      // 100 trials

    // copy back the sum from GPU
    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
