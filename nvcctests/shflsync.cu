#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>


#define REDUX_REDUCE false

// Reduction by warp using shfl.sync

__inline__ __device__
unsigned warpReduceRedux(unsigned val) {
    int laneId = threadIdx.x & 0x1f;
    unsigned reduce_element;
    unsigned remote_element;
    if (laneId == 0)
        reduce_element = val;

    remote_element = __shfl_sync(0xff, reduce_element, 0, 0x1f);
    asm volatile ("redux.sync.u32.add %0, %1, 0xff;" :
                 "+r"(remote_element) : "r"(reduce_element));

    return remote_element;
}

__inline__ __device__
unsigned warpReduceShfl(unsigned val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xff, val, offset);
    return val;
}

__inline__ __device__
unsigned blockReduce(unsigned val) {
    static __shared__ int shared[32];
    int lane = threadIdx.x%32;
    int wid = threadIdx.x / 32;
    val = warpReduceShfl(val);

    if (lane == 0)
        shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x<blockDim.x / 32) ? shared[lane] : int(0);
    if (wid == 0)
        val = REDUX_REDUCE ? warpReduceRedux(val) : warpReduceShfl(val);

    return val;
}

__global__ void reduceKernel(unsigned *in, unsigned* out, int N) {
    unsigned sum = in[threadIdx.x];
    sum = warpReduceShfl(sum);
    if (threadIdx.x == 0)
        out[blockIdx.x] = sum;
}

void deviceReduce(unsigned *in, unsigned* out, int N) {
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    int blocks = N / maxThreadsPerBlock;
    // Begin device execution
    reduceKernel<<<blocks, threads>>>(in, out, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
}


int main()
{
    const int ARRAY_SIZE = 1024;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned);

    // generate the input array on the host
    unsigned h_in[ARRAY_SIZE];
    unsigned sum = 0.0f;
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = i;
        sum += h_in[i];
    }

    // declare GPU memory pointers
    unsigned * d_in, *d_out;

    // allocate GPU memory
    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_out, sizeof(unsigned));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // offload to device
    deviceReduce(d_in, d_out, ARRAY_SIZE);

    // copy back the sum from GPU
    unsigned h_out;
    hipMemcpy(&h_out, d_out, sizeof(unsigned), hipMemcpyDeviceToHost);
    printf("%u\n", h_out);
}
