#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
#include <cudaReduce.hpp>

#define N 1000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation. Checksum: 499495.
 * Runtime values from gpu_a100 JLSE machine. 
 * Approx. runtimes: Total: 76-82ms. Mem alloc: 76-82ms.
 * Reduction: 0.02ms. 
 */

__global__ void reduce_GPU(bar::AtomicReduction<unsigned> *d) {

    /* shared memory */
    extern __shared__ bar::AtomicScalar<unsigned> sdata[];
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x*blockDim.x + threadIdx.x;

    /* load into shared memory */
    sdata[tid] = d[0][i];


    /* reduction */
    for (unsigned s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    if (tid==0) d[0][0] = sdata[0];
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

/* CPU timing functions */
int main(int argc, char **argv) {

    /* Timing variables */
    struct timeval etstart, etstop;
    struct timezone tzdummy;
    clock_t etstart2, etstop2;
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;

    /* initialize reducer on device */
    unsigned* h;
    h = (unsigned *)malloc(N*sizeof(unsigned));
    dense(h);

    bar::AtomicReduction<unsigned>arr_h(h);
    bar::AtomicReduction<unsigned> *d;
    hipMallocManaged(&d, sizeof(arr_h));
    *d = arr_h;

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

reduce_GPU<<<(N+1023) / 1024, 1024>>>(d);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;
    printf("\nElapsed time = %g ms.\n",
           (float)(usecstop - usecstart)/(float)1000);


    printf("Checksum: ");
    d[0][0].dispScalar();

    exit(0);
}
