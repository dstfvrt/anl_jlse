#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
#include <cudaReduce.hpp>

#define N 1000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation. Checksum: 499495.
 * Runtime values from gpu_a100 JLSE machine. 
 * Approx. runtimes: Total: 76-82ms. Mem alloc: 76-82ms.
 * Reduction: 0.02ms. 
 */

__global__ void reduce_GPU(void *d) {

    bar::AtomicReduction<unsigned>arr_d(d);
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x*blockDim.x + threadIdx.x;

    /* reduction */
    for (unsigned s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            arr_d[tid] += arr_d[tid + s];
	}
	__syncthreads();
    }
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

int main(int argc, char **argv) {

    /* initialize reducer on device */
    void* h;
    hipMallocManaged(&h, N*sizeof(unsigned);

    dense(h);

    /* to kernel */
reduce_GPU<<<(N+1023) / 1024, 1024>>>(h);

    printf("checksum %u", h[0]);
    exit(0);
}
