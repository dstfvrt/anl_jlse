#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
#include <cudaReduce.hpp>

#define N 1000

__global__ void reduce_GPU(unsigned *d) {
    bar::AtomicReduction<unsigned>arr_d(d);
    arr_d[0] += 1;
}

int main(int argc, char **argv) {

    unsigned* h;
    hipMallocManaged(&h, N*sizeof(unsigned));

    /* to kernel */
    reduce_GPU<<<1, N>>>(h);
    hipDeviceSynchronize();
    printf("Checksum: %u\n", h[0]);
    exit(0);
}
