
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

#define N 1000000

__global__ void reduce_GPU(unsigned *d) {
    d[0] += 1;
    printf("tid: %u", threadIdx.x);
}

int main(int argc, char **argv) {

    unsigned* h;
    hipMallocManaged(&h, N*sizeof(unsigned));

    /* to kernel */
    reduce_GPU<<<1, 1, 1000>>>(h);
    hipDeviceSynchronize();
    printf("Checksum: %u\n", h[0]);
    exit(0);
}
