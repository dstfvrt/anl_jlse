#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
#include <cudaReduce.hpp>

#define N 1000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation. Checksum: 499495.
 * Runtime values from gpu_a100 JLSE machine. 
 * Approx. runtimes: Total: 76-82ms. Mem alloc: 76-82ms.
 * Reduction: 0.02ms. 
 */

__global__ void reduce_GPU(bar::AtomicReduction<unsigned> *d) {

    /* shared memory */
    extern __shared__ bar::AtomicScalar<unsigned> sdata[];
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x*blockDim.x + threadIdx.x;

    /* load into shared memory */
    sdata[tid] = d[0][i];


    /* reduction */
    for (unsigned s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
	    printf("reduction val is ");
	    sdata[tid].dispScalar();
	}
	__syncthreads();
    }

    if (tid==0) {
        d[0][0] = sdata[0];
        printf("Checksum: ");
        d[0][0].dispScalar();
    }
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

int main(int argc, char **argv) {

    /* initialize reducer on device */
    unsigned* h;
    h = (unsigned *)malloc(N*sizeof(unsigned));
    dense(h);

    bar::AtomicReduction<unsigned>arr_h(h);
    bar::AtomicReduction<unsigned> *d;
    hipMallocManaged(&d, sizeof(arr_h));
    *d = arr_h;

    /* to kernel */
reduce_GPU<<<(N+1023) / 1024, 1024>>>(d);

    exit(0);
}
