#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>


__inline__ __device__
unsigned warpReduceRedux(unsigned val) {
    return __nvvm_redux_sync_add(val, 0xFF);
}

__global__ void reduceKernel(unsigned *in, unsigned* out, int N) {
    unsigned sum = in[threadIdx.x];
    sum = warpReduceRedux(sum);
    if (threadIdx.x == 0)
        out[0] = sum;
}

int main()
{
    const int SIZE = 32;
    const int ARRAY_BYTES = SIZE * sizeof(unsigned);

    // generate the input array on the host
    unsigned h_in[SIZE];
    unsigned sum = 0.0f;
    for (int i = 0; i < SIZE; i++) {
        h_in[i] = i;
        sum += h_in[i];
    }

    // declare GPU memory pointers
    unsigned * d_in, *d_out;

    // allocate GPU memory
    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_out, sizeof(unsigned));

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // offload to device
    reduceKernel<<<1, SIZE>>>(d_in, d_out, SIZE);

    // copy back the sum from GPU
    unsigned h_out;
    hipMemcpy(&h_out, d_out, sizeof(unsigned), hipMemcpyDeviceToHost);
    printf("%u\n", h_out);
}
