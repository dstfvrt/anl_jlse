#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N 1000000

__global__ void reduce_GPU(unsigned* d) {

    const int tid = threadIdx.x;
    int step_size = 1;
    int thread_count = blockDim.x;

    while (thread_count > 0) {

        if (tid < thread_count) {
         
	    int l_ind = tid * step_size * 2;
	    int r_ind = l_ind + step_size;
	    d[l_ind] += d[r_ind];
        }

	step_size <<= 1;
	thread_count >>=1;
    }
}

void dense(unsigned* h) {

    srand((unsigned)time(NULL));
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

/* CPU timing functions */
int main(int argc, char **argv) {

    unsigned* h;
    unsigned result = 0;
    int nBytes;
    nBytes = N*sizeof(unsigned);

    /* Timing variables */
    struct timeval etstart, etstop;
    struct timezone tzdummy;
    clock_t etstart2, etstop2;
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;

    h = (unsigned *)malloc(nBytes);
    dense(h);

    hipMalloc(&d, sizeof(unsigned));
    hipMemcpy(d, h, sizeof(unsigned), hipMemcpyHostToDevice);

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

    reduce_GPU<<<1, N / 2>>>(d);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    int result;
    hipMemcpy(&result, d, nBytes, hipMemcpyDeviceToHost);
    printf("Checksum: %u\n", result);

    /* Display timing results */
    printf("\nElapsed time = %g ms.\n",
           (float)(usecstop - usecstart)/(float)1000);

    printf("(CPU times are accurate to the nearest %g ms)\n",
           1.0/(float)CLOCKS_PER_SEC * 1000.0);
    printf("My total CPU time for parent = %g ms.\n",
           (float)( (cputstop.tms_utime + cputstop.tms_stime) -
                    (cputstart.tms_utime + cputstart.tms_stime) ) /
           (float)CLOCKS_PER_SEC * 1000);

    exit(0);
}


/* CUDA timing functions
int main(int argc, char **argv) {
    unsigned* h; 
    unsigned* d;
    int nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    dense(h);

    hipMalloc(&d, sizeof(unsigned));
    hipMemcpy(d, h, sizeof(unsigned), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("\n------------------------------\n");
    printf("\nStarting clock.\n");

    hipEventRecord(start);
    reduce_GPU<<<1, N / 2>>>(d);
    hipEventRecord(stop);
    printf("\nStopping clock.\n");

    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    int result;
    hipMemcpy(&result, d, nBytes, hipMemcpyDeviceToHost);
    printf("\nElapsed time: %f ms", elapsed);
    printf("\nsum: %d\n", result);

    exit(0);
}
*/
