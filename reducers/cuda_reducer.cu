
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

#define N 1000000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation.
 * Checksum: 383
 */
__global__ void reduce_GPU(unsigned* d) {
    /* Shared memory */
    extern __shared__ unsigned sdata[];

    /* load into shared memory */
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d[i];
    __syncthreads();

    /* reduction */
    for (unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    if (tid==0) d[0] = sdata[0];
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

/* CPU timing functions */
int main(int argc, char **argv) {

    unsigned* h;
    unsigned* d;
    unsigned result;
    unsigned nBytes;
    nBytes = N*sizeof(unsigned);

    /* Timing variables */
    struct timeval etstart, etstop;
    struct timezone tzdummy;
    clock_t etstart2, etstop2;
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;

    h = (unsigned *)malloc(nBytes);
    dense(h);

    hipMalloc(&d, nBytes);
    hipMemcpy(d, h, nBytes, hipMemcpyHostToDevice);

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

    reduce_GPU<<<1, N / 2>>>(d);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    hipMemcpy(&result, d, sizeof(unsigned), hipMemcpyDeviceToHost);
    printf("Checksum: %u\n", result);

    printf("\nElapsed time = %g ms.\n",
           (float)(usecstop - usecstart)/(float)1000);

    exit(0);
}


/* CUDA timing functions
int main(int argc, char **argv) {
    unsigned* h; 
    unsigned* d;
    int nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    dense(h);

    cudaMalloc(&d, sizeof(unsigned));
    cudaMemcpy(d, h, sizeof(unsigned), cudaMemcpyHostToDevice);
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    printf("\n------------------------------\n");
    printf("\nStarting clock.\n");

    cudaEventRecord(start);
    reduce_GPU<<<1, N / 2>>>(d);
    cudaEventRecord(stop);
    printf("\nStopping clock.\n");

    cudaEventSynchronize(stop);

    float elapsed;
    cudaEventElapsedTime(&elapsed, start, stop);
    cudaEventDestroy(start);
    cudaEventDestroy(stop);

    int result;
    cudaMemcpy(&result, d, nBytes, cudaMemcpyDeviceToHost);
    printf("\nElapsed time: %f ms", elapsed);
    printf("\nsum: %d\n", result);

    exit(0);
}
*/
