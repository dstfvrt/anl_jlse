
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
#include <omp.h>


#define N 1000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation.
 * Checksum: 490204
 */

__global__ void reduce_GPU(unsigned* d) {
    /* Shared memory */
    extern __shared__ unsigned sdata[];

    /* load into shared memory */
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d[i];
    __syncthreads();

    /* reduction */
    for (unsigned s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    if (tid==0) d[0] = sdata[0];
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

/* CPU timing functions */
int main(int argc, char **argv) {

    unsigned* h;
    unsigned* d;
    unsigned* result;
    unsigned nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    result = (unsigned *)malloc(nBytes);
    dense(h);

    /* Timing variables */
    struct timeval etstart, etstop;
    struct timezone tzdummy;
    clock_t etstart2, etstop2;
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;

    hipMalloc(&d, nBytes);
    hipMemcpy(d, h, nBytes, hipMemcpyHostToDevice);

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

    reduce_GPU<<<(N+1023) / 1024, 1024, nBytes>>>(d);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;
    printf("\nElapsed time = %g ms.\n",
           (float)(usecstop - usecstart)/(float)1000);

    hipMemcpy(result, d, nBytes, hipMemcpyDeviceToHost);
    printf("Checksum: %u\n", result[0]);

    exit(0);
}


/* CUDA timing functions
int main(int argc, char **argv) {
    unsigned* h; 
    unsigned* d;
    int nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    dense(h);

    cudaMalloc(&d, sizeof(unsigned));
    cudaMemcpy(d, h, sizeof(unsigned), cudaMemcpyHostToDevice);
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    printf("\n------------------------------\n");
    printf("\nStarting clock.\n");

    cudaEventRecord(start);
    reduce_GPU<<<1, N / 2>>>(d);
    cudaEventRecord(stop);
    printf("\nStopping clock.\n");

    cudaEventSynchronize(stop);

    float elapsed;
    cudaEventElapsedTime(&elapsed, start, stop);
    cudaEventDestroy(start);
    cudaEventDestroy(stop);

    int result;
    cudaMemcpy(&result, d, nBytes, cudaMemcpyDeviceToHost);
    printf("\nElapsed time: %f ms", elapsed);
    printf("\nsum: %d\n", result);

    exit(0);
}
*/
