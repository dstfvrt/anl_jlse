
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1000000

/* Summary: reduction for 1mil unsigned values
 * by custom CUDA implementation.
 * Checksum: 383
 */
__global__ void reduce_GPU(unsigned* d) {
    /* Shared memory */
    extern __shared__ unsigned sdata[];

    /* load into shared memory */
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = d[i];
    __syncthreads();

    /* reduction */
    for (unsigned s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    if (tid==0) d[0] = sdata[0];
}

void dense(unsigned* h) {

    srand(0);
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

/* CPU timing functions */
int main(int argc, char **argv) {

    unsigned* h;
    unsigned* d;
    unsigned result;
    unsigned nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    dense(h);

    reduce_GPU<<<(N+1023) / 1024, 1024, nBytes>>>(d);

    hipMemcpy(&result, d, nBytes, hipMemcpyDeviceToHost);
    printf("Checksum: %u\n", result);

    exit(0);
}


/* CUDA timing functions
int main(int argc, char **argv) {
    unsigned* h; 
    unsigned* d;
    int nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    dense(h);

    cudaMalloc(&d, sizeof(unsigned));
    cudaMemcpy(d, h, sizeof(unsigned), cudaMemcpyHostToDevice);
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    printf("\n------------------------------\n");
    printf("\nStarting clock.\n");

    cudaEventRecord(start);
    reduce_GPU<<<1, N / 2>>>(d);
    cudaEventRecord(stop);
    printf("\nStopping clock.\n");

    cudaEventSynchronize(stop);

    float elapsed;
    cudaEventElapsedTime(&elapsed, start, stop);
    cudaEventDestroy(start);
    cudaEventDestroy(stop);

    int result;
    cudaMemcpy(&result, d, nBytes, cudaMemcpyDeviceToHost);
    printf("\nElapsed time: %f ms", elapsed);
    printf("\nsum: %d\n", result);

    exit(0);
}
*/
