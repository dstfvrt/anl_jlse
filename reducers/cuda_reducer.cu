
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N 1000000

__global__ void reduce_GPU(unsigned* d) {

    const int tid = threadIdx.x;
    int step_size = 1;
    int thread_count = blockDim.x;

    while (thread_count > 0) {

        if (tid < thread_count) {
         
	    int l_ind = tid * step_size * 2;
	    int r_ind = l_ind + step_size;
	    d[l_ind] += d[r_ind];
        }

	step_size <<= 1;
	thread_count >>=1;
    }
}

void dense(unsigned* h) {

    srand((unsigned)time(NULL));
    for (unsigned i = 0; i < N; i++) {
        h[i] = (unsigned)rand() % 1000;
    }
}

int main(int argc, char **argv) {
    unsigned* h; 
    unsigned* d;
    int nBytes;
    nBytes = N*sizeof(unsigned);

    h = (unsigned *)malloc(nBytes);
    dense(h);

    hipMalloc(&d, sizeof(unsigned));
    hipMemcpy(d, h, sizeof(unsigned), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("\n------------------------------\n");
    printf("\nStarting clock.\n");

    hipEventRecord(start);
    reduce_GPU<<<1, N / 2>>>(d);
    hipEventRecord(stop);
    printf("\nStopping clock.\n");

    hipEventSynchronize(stop);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    int result;
    hipMemcpy(&result, d, nBytes, hipMemcpyDeviceToHost);
    printf("\nElapsed time: %f ms", elapsed);
    printf("\nsum: %d\n", result);

    exit(0);
}
